#include "hip/hip_runtime.h"
//
// Created by jacob on 9/23/24.
//
#include <stdio.h>
#include "gpu_matrix_multiply.h"
#include "hip/hip_runtime.h"

// CUDA kernel function for matrix multiplication
template <typename T>
__global__ void matrixMultiplyKernel(T *A, T *B, T *C, int l, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < l && col < n) {
        int result = 0;
        for (int i = 0; i < m; i++) {
            result += A[row * m + i] * B[i * n + col];
        }
        C[row * n + col] = result;
    }
}

template <typename T>
__host__ void matrixMultiply(matrix_2d *A, matrix_2d *B, matrix_2d *C) {
    // number of columns in matrix A (row) and rows in matrix B
    int m = A->x_length;
    // number of rows (column) in matrix A
    int l = A->y_length;
    // number of columns in matrix B
    int n = B->x_length;

    int A_size = A->x_length * A->y_length *  sizeof(T);
    int B_size = B->x_length * B->y_length * sizeof(T);
    int C_size = C->x_length * C->y_length * sizeof(T);

    T *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, A_size);
    hipMalloc((void **) &d_B, B_size);
    hipMalloc((void**) &d_C, C_size);

    hipMemcpy(d_A, A->data, A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B->data, B_size, hipMemcpyHostToDevice);

    // matrix multiply
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (l + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMultiplyKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, l, m, n);

    hipDeviceSynchronize();

    hipMemcpy(C->data, d_C, C_size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__host__ void matrix2D_gpu_matmul(matrix_2d *A, matrix_2d *B, matrix_2d *C) {
    if (A->precision == INT) {
        matrixMultiply<int>(A, B, C);
    } else {
        matrixMultiply<double>(A, B, C);
    }
}